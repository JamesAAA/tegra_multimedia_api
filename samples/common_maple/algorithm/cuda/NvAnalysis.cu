#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <hip/hip_runtime.h>
#include "NvAnalysis.h"

#define BOX_W 32
#define BOX_H 32

__global__ void
addLabelsKernel(int *pDevPtr, int pitch)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y + BOX_H;
    int col = blockIdx.x * blockDim.x + threadIdx.x + BOX_W;
    char *pElement = (char *)pDevPtr + row * pitch + col;

    pElement[0] = 0;

    return;
}

int 
addLabels(hipDeviceptr_t pDevPtr, int pitch)
{
    dim3 threadsPerBlock(BOX_W, BOX_H);
    dim3 blocks(1,1);

    addLabelsKernel<<<blocks,threadsPerBlock>>>((int *)pDevPtr, pitch);

    return 0;
}

__global__ void
convertIntToFloatKernelRGB(hipDeviceptr_t pDevPtr, int width, int height,
                void* cuda_buf, int pitch)
{
    float *pdata = (float *)cuda_buf;
    char *psrcdata = (char *)pDevPtr;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (col < width && row < height)
    {
        for (int k = 0; k < 3; k++)
        {
            pdata[width * height * k + row * width + col] =
                ((float)*(psrcdata + row * pitch + col * 4 + (3 - 1 - k))/255);
        }
    }
}

__global__ void
convertIntToFloatKernelBGR(hipDeviceptr_t pDevPtr, int width, int height,
                void* cuda_buf, int pitch)
{
    float *pdata = (float *)cuda_buf;
    char *psrcdata = (char *)pDevPtr;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // BGR offset for 3 classes
    //int offsets[] = {124, 117, 104};

    if (col < width && row < height)
    {
        // For V4L2_PIX_FMT_ABGR32 --> BGRA-8-8-8-8
        for (int k = 0; k < 3; k++)
        {
            pdata[width * height * k + row * width + col] =
                ((float)*(psrcdata + row * pitch + col * 4 + k)/255);
        }
    }
}

__global__ void
convertRGBtoABGR(void* pDevPtr, int width, int height,
                void* cuda_buf)
{
    char *pdata = (char *)cuda_buf;
    char *psrcdata = (char *)pDevPtr;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (col < width && row < height)
    {
        for (int k = 0; k < 3; k++)
        {
            pdata[row * width * 4 + col * 4 + (2-k)] =
                *(psrcdata + row * width * 3 + col * 3 + k);
        }
		pdata[row * width * 4 + col * 4 + 3] = 255;
    }
}

__global__ void
convertBGRtoABGR(void* pDevPtr, int width, int height,
                void* cuda_buf)
{
    char *pdata = (char *)cuda_buf;
    char *psrcdata = (char *)pDevPtr;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (col < width && row < height)
    {
        for (int k = 0; k < 3; k++)
        {
            pdata[row * width *4 + col * 4 + k] =
                *(psrcdata + row * width * 3 + col * 3 + k);
        }
		pdata[row * width * 4 + col * 4 + 3] = 255;
    }
}

__global__ void
convertARGBtoABGR(void* pDevPtr, int width, int height,
                void* cuda_buf)
{
    char *pdata = (char *)cuda_buf;
    char *psrcdata = (char *)pDevPtr;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (col < width && row < height)
    {
        for (int k = 0; k < 3; k++)
        {
            pdata[row * width * 4 + col * 4 + (3 - k)] =
                *(psrcdata + row * width * 4 + col * 4 + k);
        }
		//pdata[row * width * 4 + col * 4 + 3] = 255;
    }
}

__global__ void
convertYUY2toI420(void* pDevPtr, int width, int height,
                void* cuda_buf)
{
    char *pdata = (char *)cuda_buf;
    char *psrcdata = (char *)pDevPtr;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (col < width && row < height)
    {
        if(col % 2 == 0)  // Y
            pdata[row * width / 2 + (col / 2)] = *(psrcdata + row * width + col);
        else if((row % 2 == 0) && (col % 4 == 1))    // U
            pdata[width * height / 2 + (row / 2 ) * (width/4) + (col / 4)] = *(psrcdata + row * width + col);
        else if((row % 2 == 0) && (col % 4 == 3))    // V
            pdata[(width * height * 5 / 8) + (row / 2 ) * (width/4) + (col / 4)] = *(psrcdata + row * width + col);        
    } 
}

int convertFormat(void* pDevPtr,
                      int width,
                      int height,
                      COLOR_FORMAT color_format,
                      void* cuda_buf)
{
    dim3 threadsPerBlock(32, 32);
    dim3 blocks((width+31)/threadsPerBlock.x, (height+31)/threadsPerBlock.y);

	if (color_format == COLOR_FORMAT_ARGBTOABGR)
    {
        convertARGBtoABGR<<<blocks, threadsPerBlock>>>(pDevPtr, width,
                height, cuda_buf);
    }
    else if (color_format == COLOR_FORMAT_BGRTOABGR)
    {
        convertBGRtoABGR<<<blocks, threadsPerBlock>>>(pDevPtr, width,
                height, cuda_buf);
    }
    else if (color_format == COLOR_FORMAT_RGBTOABGR)
    {
        convertRGBtoABGR<<<blocks, threadsPerBlock>>>(pDevPtr, width,
                height, cuda_buf);
    }
    else if (color_format == COLOR_FORMAT_YUY2TOI420)
    {
        convertYUY2toI420<<<blocks, threadsPerBlock>>>(pDevPtr, width,
                height, cuda_buf);
    }    

    return 0;
}


int convertIntToFloat(hipDeviceptr_t pDevPtr,
                      int width,
                      int height,
                      int pitch,
                      COLOR_FORMAT color_format,
                      void* cuda_buf)
{
    dim3 threadsPerBlock(32, 32);
    dim3 blocks(width/threadsPerBlock.x, height/threadsPerBlock.y);

    if (color_format == COLOR_FORMAT_RGB)
    {
        convertIntToFloatKernelRGB<<<blocks, threadsPerBlock>>>(pDevPtr, width,
                height, cuda_buf, pitch);
    }
    else if (color_format == COLOR_FORMAT_BGR)
    {
        convertIntToFloatKernelBGR<<<blocks, threadsPerBlock>>>(pDevPtr, width,
                height, cuda_buf, pitch);
    }

    return 0;
}
